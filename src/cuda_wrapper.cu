#include "hip/hip_runtime.h"
#include "cuda_wrapper.cuh"



template class thrust::device_vector<double>;
template class thrust::device_ptr<double>;
template thrust::device_ptr<double> thrust::for_each_n(const thrust::detail::execution_policy_base<thrust::cuda_cub::tag> &, thrust::device_ptr<double>, unsigned long, thrust::detail::device_generate_functor<thrust::detail::fill_functor<double>>);
template eigen_iterator thrust::copy(const thrust::detail::execution_policy_base<thrust::cuda_cub::cross_system<thrust::cuda_cub::tag, thrust::system::cpp::detail::tag>> &, thrust_const_iterator, thrust_const_iterator, eigen_iterator);

template thrust_iterator thrust::copy(eigen_iterator, eigen_iterator, thrust_iterator);
template eigen_iterator thrust::copy(thrust_iterator, thrust_iterator, eigen_iterator);

/*
  This code doesn't work as I intended it to.
  I want to be able to call this function in translation units compiled by g++/icpx.
  However, whenever I call a Eigen::VectorXd constructor compiled by nvcc and call its destructor compiled by g++/icpx, I get a segfault.
  Calling the constructor/destructor compiled by the same compiler is okay though.
*/
Eigen::VectorXd copy_vector(const thrust::device_vector<double> &in)
{
  //Eigen::VectorXd out(static_cast<long long int>(in.size()));
  //Eigen::VectorXd out = Eigen::VectorXd::Zero(in.size());
  Eigen::VectorXd out;
  out.resize(static_cast<long long int>(in.size()));
  out.array() = 0;
  std::cout << "out.size() = " << out.size() << '\n';
  hipMemcpy((void *)out.data(), (const void *)thrust::raw_pointer_cast(in.data()), in.size() * sizeof(double), hipMemcpyDeviceToHost);
  // std::cout << "error = " << error << '\n';
  // std::cout << "in.size() = " << in.size() << '\n';
  return out;
}

/*
  Same issue as above.
*/
Eigen::VectorXd copy_vector(const Eigen::VectorXd &in)
{
  return Eigen::VectorXd(in);
}

/*
  This code works, but now Eigen::VectorXd &out has to be allocated outside this function.
  (Typically in a translation unit compiled by g++/icpx.)
*/
void copy_vector(Eigen::VectorXd &out, const thrust::device_vector<double> &in)
{
  assert(out.size() >= in.size());
  
  //thrust::copy(in.begin(), in.end(), out.begin());  
  hipMemcpy((void *)out.data(), (const void *)thrust::raw_pointer_cast(in.data()), in.size() * sizeof(double), hipMemcpyDeviceToHost);
  
  // No need to synchronize. For transfers from device to either pageable or pinned host memory, the function returns only once the copy has completed. See https://docs.nvidia.com/cuda/cuda-driver-api/api-sync-behavior.html.
  // hipStreamSynchronize(0);
}

// void copy_vector(Eigen::VectorXd &out, const Eigen::VectorXd &in)
// {
//   out = in;
// }

void show_gpu_memory_usage(void)
{
  size_t free, total;
  hipMemGetInfo(&free, &total);
  std::cout << "free / total = "
	    << free << " B / " << total << " B ("
	    << free / (1024*1024) << " MB / " << total / (1024*1024) << " MB)\n";
}

cufftWrapperD2Z::cufftWrapperD2Z(int N_) : N(N_)
{
  hipfftPlan3d(&plan, N_, N_, N_, HIPFFT_D2Z);
  //std::cout << "plan initialized!\n";
}  

cufftWrapperD2Z::~cufftWrapperD2Z()
{
  hipfftDestroy(plan);
  //std::cout << "plan destoyed!\n";
}

thrust::device_vector<double> cufftWrapperD2Z::execute(thrust::device_vector<double> &in)
{
  //std::cout << "executing plan!\n";
  thrust::device_vector<double> out(N * N * (N / 2 + 1) * 2);
  hipfftExecD2Z(plan, thrust::raw_pointer_cast(in.data()), (hipfftDoubleComplex *)thrust::raw_pointer_cast(out.data()));
  return out;
}


cufftWrapperBatchedD2Z::cufftWrapperBatchedD2Z(int N_) : N(N_)
{
  int rank = 3;
  int n[3] = {N_, N_, N_};
  int batch = 2;
  hipfftPlanMany(&plan, rank, n, NULL, 0, 0, NULL, 0, 0, HIPFFT_D2Z, batch);
  // std::cout << "plan initialized!\n";
  // size_t workSize;
  // hipfftGetSize(plan, &workSize);
  // std::cout << "workSize = " << workSize << '\n';
}  

cufftWrapperBatchedD2Z::~cufftWrapperBatchedD2Z()
{
  hipfftDestroy(plan);
  // std::cout << "plan destoyed!\n";
}

thrust::device_vector<double> cufftWrapperBatchedD2Z::execute(thrust::device_vector<double> &in)
{
  // std::cout << "executing plan!\n";
  thrust::device_vector<double> out(N * N * (N / 2 + 1) * 2 * 2);
  hipfftExecD2Z(plan, thrust::raw_pointer_cast(in.data()), (hipfftDoubleComplex *)thrust::raw_pointer_cast(out.data()));
  return out;
}


__device__
hipfftDoubleComplex scale_callback(void *dataIn, size_t offset, void *callerInfo, void *sharedPtr) {
  int N = 384;
  int N3 = N * N * N;
  double r = ((hipfftDoubleComplex *)dataIn)[offset].x / N3;
  double i = ((hipfftDoubleComplex *)dataIn)[offset].y / N3;
  return make_hipDoubleComplex(r, i);
}

__device__ hipfftCallbackLoadZ scale_callback_ptr = scale_callback;
hipfftCallbackLoadZ hostCopyOfCallbackPtr;

cufftWrapper::cufftWrapper(int N_) : N(N_)
{
  hipfftCreate(&plan_d2z);
  hipfftCreate(&plan_batched_d2z);
  hipfftCreate(&plan_z2d);

  hipfftSetAutoAllocation(plan_d2z, 0);
  hipfftSetAutoAllocation(plan_batched_d2z, 0);
  hipfftSetAutoAllocation(plan_z2d, 0);

  size_t workSize_d2z, workSize_batched_d2z, workSize_z2d;

  hipfftMakePlan3d(plan_d2z, N_, N_, N_, HIPFFT_D2Z, &workSize_d2z);

  int rank = 3;
  int n[3] = {N_, N_, N_};
  int batch = 2;
  hipfftMakePlanMany(plan_batched_d2z, rank, n, NULL, 0, 0, NULL, 0, 0, HIPFFT_D2Z, batch, &workSize_batched_d2z);

  
  hipfftMakePlan3d(plan_z2d, N_, N_, N_, HIPFFT_Z2D, &workSize_z2d);
 
  size_t required_size = std::max({workSize_d2z, workSize_batched_d2z, workSize_z2d});
 
  work_area.resize(required_size / sizeof(double));
  
  //std::cout << "required_sizes = " << workSize_d2z << ", "
  //	    << workSize_batched_d2z << ", "
  //	    << workSize_z2d << '\n';
  //std::cout << "max required_size = " << required_size << '\n';
  //show_gpu_memory_usage();

  hipfftSetWorkArea(plan_d2z, thrust::raw_pointer_cast(work_area.data()));
  hipfftSetWorkArea(plan_batched_d2z, thrust::raw_pointer_cast(work_area.data()));
  hipfftSetWorkArea(plan_z2d, thrust::raw_pointer_cast(work_area.data()));

  // hipMemcpyFromSymbol(&hostCopyOfCallbackPtr, HIP_SYMBOL(scale_callback_ptr), sizeof(hostCopyOfCallbackPtr));
  // hipfftResult result = hipfftXtSetCallback(plan_z2d, (void **)&hostCopyOfCallbackPtr, HIPFFT_CB_LD_COMPLEX_DOUBLE, NULL);
  // std::cout << "no error = " << (result == HIPFFT_SUCCESS) << '\n';
  // std::cout << "result = " << result << '\n';
}  

cufftWrapper::~cufftWrapper()
{
  hipfftDestroy(plan_d2z);
  hipfftDestroy(plan_batched_d2z);
  hipfftDestroy(plan_z2d);
}

thrust::device_vector<double> cufftWrapper::execute_d2z(thrust::device_vector<double> &in)
{
  thrust::device_vector<double> out(N * N * (N / 2 + 1) * 2);
  hipfftExecD2Z(plan_d2z, thrust::raw_pointer_cast(in.data()), (hipfftDoubleComplex *)thrust::raw_pointer_cast(out.data()));
  return out;
}

thrust::device_vector<double> cufftWrapper::execute_batched_d2z(thrust::device_vector<double> &in)
{
  thrust::device_vector<double> out(N * N * (N / 2 + 1) * 2 * 2);
  hipfftExecD2Z(plan_batched_d2z, thrust::raw_pointer_cast(in.data()), (hipfftDoubleComplex *)thrust::raw_pointer_cast(out.data()));
  return out;
}

thrust::device_vector<double> cufftWrapper::execute_z2d(thrust::device_vector<double> &in)
{
  thrust::device_vector<double> out(N * N * N);
  hipfftExecZ2D(plan_z2d, (hipfftDoubleComplex *)thrust::raw_pointer_cast(in.data()), thrust::raw_pointer_cast(out.data()));
  return out;
}



cufftWrapperNoBatching::cufftWrapperNoBatching(int N_) : N(N_)
{
  hipfftCreate(&plan_d2z);
  hipfftCreate(&plan_z2d);

  hipfftSetAutoAllocation(plan_d2z, 0);
  hipfftSetAutoAllocation(plan_z2d, 0);

  size_t workSize_d2z, workSize_z2d;

  hipfftMakePlan3d(plan_d2z, N_, N_, N_, HIPFFT_D2Z, &workSize_d2z);

  hipfftMakePlan3d(plan_z2d, N_, N_, N_, HIPFFT_Z2D, &workSize_z2d);
 
  size_t required_size = std::max({workSize_d2z, workSize_z2d});
 
  work_area.resize(required_size / sizeof(double));
  
  hipfftSetWorkArea(plan_d2z, thrust::raw_pointer_cast(work_area.data()));
  hipfftSetWorkArea(plan_z2d, thrust::raw_pointer_cast(work_area.data()));
}  

cufftWrapperNoBatching::~cufftWrapperNoBatching()
{
  hipfftDestroy(plan_d2z);
  hipfftDestroy(plan_z2d);
}

thrust::device_vector<double> cufftWrapperNoBatching::execute_d2z(thrust::device_vector<double> &in)
{
  thrust::device_vector<double> out(N * N * (N / 2 + 1) * 2);
  hipfftExecD2Z(plan_d2z, thrust::raw_pointer_cast(in.data()), (hipfftDoubleComplex *)thrust::raw_pointer_cast(out.data()));
  return out;
}

thrust::device_vector<double> cufftWrapperNoBatching::execute_batched_d2z(thrust::device_vector<double> &in)
{
  thrust::device_vector<double> out(N * N * (N / 2 + 1) * 2 * 2);
  hipfftExecD2Z(plan_d2z, thrust::raw_pointer_cast(in.data()), (hipfftDoubleComplex *)thrust::raw_pointer_cast(out.data()));
  hipfftExecD2Z(plan_d2z, thrust::raw_pointer_cast(in.data()) + N*N*N, ((hipfftDoubleComplex *)thrust::raw_pointer_cast(out.data())) + N*N*(N/2+1));
  return out;
}

thrust::device_vector<double> cufftWrapperNoBatching::execute_z2d(thrust::device_vector<double> &in)
{
  thrust::device_vector<double> out(N * N * N);
  hipfftExecZ2D(plan_z2d, (hipfftDoubleComplex *)thrust::raw_pointer_cast(in.data()), thrust::raw_pointer_cast(out.data()));
  return out;
}

void cufftWrapperNoBatching::execute_inplace_z2d(thrust::device_vector<double> &inout)
{
  hipfftExecZ2D(plan_z2d, (hipfftDoubleComplex *)thrust::raw_pointer_cast(inout.data()), thrust::raw_pointer_cast(inout.data()));
}

/*
// Simple transformation kernel
__global__ void transformKernel(float* output,
                                hipTextureObject_t texObj,
                                int width, int height,
                                float theta)
{
    // Calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = x / (float)width;
    float v = y / (float)height;

    // Transform coordinates
    u -= 0.5f;
    v -= 0.5f;
    float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
    float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

    // Read from texture and write to global memory
    output[y * width + x] = tex2D<float>(texObj, tu, tv);
}
*/

 // Another transformation kernel
__global__ void transformKernel(float* output,
                                hipTextureObject_t texObj)
{
  // Calculate normalized texture coordinates
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

  float nx = (x+1) / 16.0;
  
  // Read from texture and write to global memory
  //output[x] = tex1Dfetch<float>(texObj, x);
  output[x] = tex1D<float>(texObj, nx);
  //output[x] = x;
}



void test_texture(void)
{
  int N = 8;
  // size_t size = N * sizeof(float);
  // float *h_data = (float *)std::malloc(size);
  // for (int i = 0; i < N; ++i) {
  //   h_data[i] = std::cos(i * 2 * std::numbers::pi / N);
  // }
  // float *dData = NULL;
  // checkCudaErrors(hipMalloc((void **) &dData, size));

  Eigen::VectorXf vec_h(N);
  thrust::device_vector<float> vec_d(N);
  
  for(int i = 0; i < N; ++i) {
    vec_h[i] = std::cos(i * 2 * std::numbers::pi / N);
  }
  thrust::copy(vec_h.begin(), vec_h.end(), vec_d.begin());
  

  hipChannelFormatDesc channelDesc =
    hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  
  hipTextureObject_t texture_obj = 0;
  hipResourceDesc resource_desc;
  hipTextureDesc texture_desc;
  //hipResourceViewDesc resource_view_desc;
  
  // resource_desc.resType = hipResourceTypeLinear;
  // resource_desc.res.linear.devPtr = (void *)thrust::raw_pointer_cast(vec_d.data());
  // resource_desc.res.linear.desc = channelDesc;
  // resource_desc.res.linear.sizeInBytes = vec_d.size() * sizeof(float);
  hipArray_t cuArray;
  //hipMallocArray(&cuArray, &channelDesc, N);
  hipMallocArray(&cuArray, &channelDesc, N);
  hipMemcpyToArray(cuArray, 0, 0, vec_h.data(),
		    sizeof(float) * N, hipMemcpyHostToDevice);
  

  // Specify texture
  memset(&resource_desc, 0, sizeof(resource_desc));
  resource_desc.resType = hipResourceTypeArray;
  resource_desc.res.array.array = cuArray;


  // Specify texture object parameters
  memset(&texture_desc, 0, sizeof(texture_desc));
  texture_desc.addressMode[0] = hipAddressModeWrap;
  //texture_desc.addressMode[1] = hipAddressModeWrap;
  texture_desc.filterMode = hipFilterModeLinear;
  texture_desc.readMode = hipReadModeElementType;
  texture_desc.normalizedCoords = 1;

  //hipBindTextureToArray(&texure_desc, cuArray);
  
  // Create texture object
  hipCreateTextureObject(&texture_obj, &resource_desc, &texture_desc, NULL);

  
  Eigen::VectorXf out_h(2*N);
  thrust::device_vector<float> out_d(2*N);

  
  // Invoke kernel
  dim3 threadsperBlock(N);
  dim3 numBlocks(2);
  transformKernel<<<numBlocks, threadsperBlock>>>(thrust::raw_pointer_cast(out_d.data()), texture_obj);
  
  
  // Copy data from device back to host
  //thrust::copy(out_d.begin(), out_d.end(), out_h.begin());  
  hipMemcpy((void *)out_h.data(), (const void *)thrust::raw_pointer_cast(out_d.data()), out_d.size() * sizeof(float), hipMemcpyDeviceToHost);
  
  
  // Destroy texture object
  hipDestroyTextureObject(texture_obj);

  std::cout << "in = " << vec_h.transpose() << '\n';
  std::cout << "out = " << out_h.transpose() << '\n';

  
  /*
  // Allocate array and copy image data
  hipChannelFormatDesc channelDesc =
  hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipArray *cuArray;
  checkCudaErrors(hipMallocArray(&cuArray,
				  &channelDesc,
				  width,
				  height));
  checkCudaErrors(hipMemcpyToArray(cuArray,
  0,
  0,
  hData,
  size,
  hipMemcpyHostToDevice));

  // Bind the array to the texture
  //checkCudaErrors(hipBindTextureToArray(tex, cuArray, channelDesc));

  */

}

